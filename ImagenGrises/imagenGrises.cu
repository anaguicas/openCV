#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RED 2
#define GREEN 1
#define BLUE 0

#define MASK_WIDTH 3

using namespace cv;

//---------------Función paralela----------------------------
__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 \
                                     + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}

//---------------Función secuencial------------------------
unsigned char *imagenGrises(unsigned char *gray, int height, int width, Mat image){ 
    unsigned char *imagen_salida;
    for(int i=0; i<height; i++){
        for(int j=0; j<width; j++){
        gray[(i*width+j)]= 0.299*image.data[(i*width+j)*3+2] + 0.587*image.data[(i*width+j)*3+1] + 0.114*image.data[(i*width+j)*3];
    }    
    }
    imagen_salida=gray;
    return imagen_salida;
}

int main(int argc, char **argv)
{
    hipError_t error = hipSuccess;
    clock_t start, end, startGPU, endGPU;
    double cpu_time_used, gpu_time_used;
    char* imageName = argv[1];
    unsigned char *image_aux,*gray;
    unsigned char *h_imageInput, *d_imageInput, *d_imageOutput, *h_imageOutput;
    unsigned char *h_sobelOuput,*d_sobelOutput;
    Mat image;
    image = imread(imageName, 1);

    if(argc !=2 || !image.data){
        printf("No image Data \n");
        return -1;
    }

    Size s = image.size();

    int width = s.width;
    int height = s.height;
    int size = sizeof(unsigned char)*width*height*image.channels();
    int tama = sizeof(unsigned char)*width*height;

    //------------Imagenes en escala de grises secuencial----
    Mat image_gray;

    gray = (unsigned char*)malloc(size);
    start = clock();
    image_aux=imagenGrises(gray,height,width,image);
    end=clock();
    image_gray.create(height,width,CV_8UC1);
    image_gray.data=image_aux;

    //--------------------------------------------------

    //----------------Imagen en grises paralelo---------------------
    h_imageInput = (unsigned char*)malloc(size);
    error = hipMalloc((void**)&d_imageInput,size);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_imageInput\n");
        exit(-1);
    }

    h_imageOutput = (unsigned char *)malloc(tama);
    error = hipMalloc((void**)&d_imageOutput,tama);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_imageOutput\n");
        exit(-1);
    }

    h_sobelOuput = (unsigned char*)malloc(tama);
    error= hipMalloc((void**)&d_sobelOutput,tama);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_sobelOutput\n");
        exit(-1);
    }

    h_imageInput = image.data;

    startGPU = clock();
    error = hipMemcpy(d_imageInput,h_imageInput,size, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de h_imageInput a d_imageInput \n");
        exit(-1);
    }

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);
    img2gray<<<dimGrid,dimBlock>>>(d_imageInput,width,height,d_imageOutput);
    hipDeviceSynchronize();    
    hipMemcpy(h_imageOutput,d_imageOutput,tama,hipMemcpyDeviceToHost);
       
    endGPU = clock();
 
    Mat gray_image;
    gray_image.create(height,width,CV_8UC1);
    gray_image.data = h_imageOutput;
    //---------------------------------------------------------------  

    Mat gray_image_opencv;
    cvtColor(image, gray_image_opencv, CV_BGR2GRAY);


    //imwrite("./Gray_Image.jpg",gray_image);

    namedWindow("Image", WINDOW_NORMAL);
    namedWindow("Gray Image CUDA secuencial", WINDOW_NORMAL);
    namedWindow("Gray Image CUDA paralelo", WINDOW_NORMAL);

    imshow("Image",image);
    imshow("Gray Image CUDA secuencial", image_gray);
    imshow("Gray Image CUDA paralelo", gray_image);

    waitKey(0);

    //free(h_imageInput);
    gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo Paralelo: %.10f\n",gpu_time_used);
    cpu_time_used = ((double) (end - start)) /CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo secuencial: %.10f\n",cpu_time_used);
    printf("La aceleración obtenida es de %.10fX\n",cpu_time_used/gpu_time_used);

    hipFree(d_imageInput);
    hipFree(d_imageOutput);
    return 0;
}
